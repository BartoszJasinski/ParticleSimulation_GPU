#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "ParticleSystem.h"

__host__ hipError_t simulateParticles(int);

__global__ void simulateParticles()
{
	

}

int main()
{

	// Add vectors in parallel.
	hipError_t cudaStatus = simulateParticles(1);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

int ParticleDims(ParticleSystem particle_system)
{
	return(6 * particle_system->number_of_particles);
};

int ParticleGetState(ParticleSystem particle_system, float *dst)
{
	for (int i = 0; i < particle_system->number_of_particles; i++)
	{
		*(dst++) = particle_system->particle[i]->p.x;
		*(dst++) = particle_system->particle[i]->p.y;
		*(dst++) = particle_system->particle[i]->p.z;
		*(dst++) = particle_system->particle[i]->v.x;
		*(dst++) = particle_system->particle[i]->v.y;
		*(dst++) = particle_system->particle[i]->v.z;
	}
}

int ParticleSetState(ParticleSystem particle_system, float *src)
{
	for (int i = 0; i < particle_system->number_of_particles; i++) {
		particle_system->particle[i]->p.x = *(src++);
		particle_system->particle[i]->p.y = *(src++);
		particle_system->particle[i]->p.z = *(src++);
		particle_system->particle[i]->v.x = *(src++);
		particle_system->particle[i]->v.y = *(src++);
		particle_system->particle[i]->v.z = *(src++);
	}
}

void Clear_Forces(ParticleSystem particle_system);

void Compute_Forces(ParticleSystem particle_system);

int ParticleDerivative(ParticleSystem particle_system, float *dst)
{
	
	Clear_Forces(particle_system);   /* zero the force accumulators */
	Compute_Forces(particle_system); /* magic force function */
	for (int i = 0; i < particle_system->number_of_particles; i++) {
		*(dst++) = particle_system->particle[i]->v.x;    /* xdot=v*/
		*(dst++) = particle_system->particle[i]->v.y;
		*(dst++) = particle_system->particle[i]->v.z;
		*(dst++) = particle_system->particle[i]->f.x / particle_system->particle[i]->m; /* vdot = f/m */
		*(dst++) = particle_system->particle[i]->f.y / particle_system->particle[i]->m;
		*(dst++) = particle_system->particle[i]->f.z / particle_system->particle[i]->m;
	}
}

void ScaleVector(float* temp1, float delta_t);

void AddVectors(float* vector1, float* vector2, float* result);

void EulerStep(ParticleSystem particle_system, float DeltaT)
{
	float *temp1, *temp2;
	ParticleDerivative(particle_system, temp1);   /* get deriv */
	ScaleVector(temp1, DeltaT);       /* scale it */
	ParticleGetState(particle_system, temp2);      /* get state */
	AddVectors(temp1, temp2, temp2);  /* add -> temp2 */
	ParticleSetState(particle_system, temp2);      /* update state */
	particle_system->t += DeltaT;                 /* update time */
}


__host__ hipError_t simulateParticles(int)
{
	//    hipError_t cudaStatus;
	//
	//    // Choose which GPU to run on, change this on a multi-GPU system.
	//    cudaStatus = hipSetDevice(0);
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	//        goto Error;
	//    }
	//
	//    // Allocate GPU buffers for three vectors (two input, one output)    .
	//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMalloc failed!");
	//        goto Error;
	//    }
	//
	//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMalloc failed!");
	//        goto Error;
	//    }
	//
	//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMalloc failed!");
	//        goto Error;
	//    }
	//
	//    // Copy input vectors from host memory to GPU buffers.
	//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMemcpy failed!");
	//        goto Error;
	//    }
	//
	//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMemcpy failed!");
	//        goto Error;
	//    }
	//
	//    // Launch a kernel on the GPU with one thread for each element.
	//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
	//
	//    // Check for any errors launching the kernel
	//    cudaStatus = hipGetLastError();
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "addKernel launch failed: %s\number_of_particles", hipGetErrorString(cudaStatus));
	//        goto Error;
	//    }
	//    
	//    // hipDeviceSynchronize waits for the kernel to finish, and returns
	//    // any errors encountered during the launch.
	//    cudaStatus = hipDeviceSynchronize();
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\number_of_particles", cudaStatus);
	//        goto Error;
	//    }
	//
	//    // Copy output vector from GPU buffer to host memory.
	//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	//    if (cudaStatus != hipSuccess) {
	//        fprintf(stderr, "hipMemcpy failed!");
	//        goto Error;
	//    }
	//
	//Error:
	//    hipFree(dev_c);
	//    hipFree(dev_a);
	//    hipFree(dev_b);
	//    
	//    return cudaStatus;
}
